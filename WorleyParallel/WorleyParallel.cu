#include "hip/hip_runtime.h"
//============================================================================
// Name        : WorleySerial.cpp
// Author      : Gerard Tabone
// Version     :
// Copyright   : Your copyright notice
// Description : Hello World in C++, Ansi-style
//============================================================================

#include <iostream>
#include "jbutil.h"
#include <stdio.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__
void normDistanceFromNearestPoint(int *result, float width, float height, int *random_points, int N) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if(x < width && y < height) {
		int shortest_norm_dist = 100;

		for(unsigned int i = 0; i < N; i++) {
			float x_point = random_points[i * 2];
			float y_point = random_points[i * 2 + 1];
			float x_dist = (x - x_point) / 2.0;
			float y_dist = (y - y_point) / 2.0;

			int distance = sqrt(x_dist * x_dist + y_dist * y_dist);

			shortest_norm_dist = distance < shortest_norm_dist ? distance : shortest_norm_dist;
		}

		result[x + (int) width * y] = shortest_norm_dist;
	}
}


//        x_dist = (pixel_x - point_x) / (img_width / 4)
//        y_dist = (pixel_y - point_y) / (img_height / 4)
//        norm_dist = math.sqrt(x_dist ** 2 + y_dist ** 2)

template <class real>
void process(const std::string infile, const std::string outfile,
  const real R, const int a)
{
	// start timer
	double t = jbutil::gettime();


	int height = 4000, width = 4000;
	int N = 100;
	int seed = 5;

	jbutil::randgen rand(seed);

	int random_points[N][2], *result;
	result = (int *) malloc(height * width * sizeof(int));


	for(int i = 0; i <  N; i++) {
		rand.advance();
		random_points[i][0] = (int) rand.fval(0, width);
		rand.advance();
		random_points[i][1] = (int) rand.fval(0, height);

	}

	jbutil::image<int> image_out = jbutil::image<int>(height, width, 1, 255);


	int *d_random_points, *d_result;
	size_t random_points_size = N * 2 * sizeof(int);

	gpuErrchk( hipMalloc((void**) &d_random_points, random_points_size) );
	gpuErrchk( hipMalloc((void**) &d_result, width * height * sizeof(int)) );
	gpuErrchk( hipMemcpy(d_random_points, (int *) random_points, random_points_size, hipMemcpyHostToDevice) );

	dim3 grid((width + 32 - 1) / 32, (height + 32 - 1) / 32);
	dim3 blocks(32, 32);
	normDistanceFromNearestPoint<<<grid, blocks>>>(d_result, width, height, d_random_points, N);

	gpuErrchk( hipMemcpy(result, d_result, height * width * sizeof(int) , hipMemcpyDeviceToHost) );

	hipDeviceSynchronize();

	for(unsigned int i = 0; i < width; i++) {
		for(unsigned int j = 0; j < height; j++) {
			image_out(0, i, j) = result[i + j * width];
		}
	}

//	for(int i = 0; i < height; i++) {
//	   for(int j = 0; j < width; j++) {
//		   image_out(0, i, j) = normDistanceFromNearestPoint(j, i, width, height, (int *) random_points, N)  % 255;
//	   }
//	}

	// stop timer
	t = jbutil::gettime() - t;
	// save image
	std::ofstream file_out(outfile.c_str());
	image_out.save(file_out);
	// show time taken
	std::cerr << "Time taken: " << t << "s" << std::endl;
}

// Main program entry point

int main(int argc, char *argv[])
{
	//	std::cerr << "Lab 2: Image resampling with Lanczos filter" << std::endl;
	//	if (argc != 5)
	//	{
	//		std::cerr << "Usage: " << argv[0]
	//		<< " <infile> <outfile> <scale-factor> <limit>" << std::endl;
	//		exit(1);
	//	}
	//	process<float> (argv[1], argv[2], atof(argv[3]), atoi(argv[4]));
	process<float> ("", "out.pgm", 0.1, 1);


}

